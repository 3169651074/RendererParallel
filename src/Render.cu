#include "hip/hip_runtime.h"
#include <Render.cuh>
using namespace std;

namespace renderer {
    void Renderer::setDirectSampleObject(const std::pair<PrimitiveType, size_t> * objectList, size_t objectListSize) {
        SDL_Log("Set direct sample object list.");
        if (objectList == nullptr) return;
        if (dev_spheres == nullptr && dev_parallelograms == nullptr) {
            SDL_Log("Direct sample object list not initialized, call commitSceneData first.");
            return;
        }

        size_t sphereCount = 0, parallelogramCount = 0;
        //统计数量
        for (size_t i = 0; i < objectListSize; i++) {
            switch (objectList[i].first) {
                case PrimitiveType::SPHERE:
                    sphereCount++;
                    break;
                case PrimitiveType::PARALLELOGRAM:
                    parallelogramCount++;
                    break;
                default:;
            }
        }

        //分配显存
        cudaCheckError(hipMalloc(&dev_hittablePDFSpheres, sphereCount * sizeof(Sphere *)));
        cudaCheckError(hipMalloc(&dev_hittablePDFParallelograms, parallelogramCount * sizeof(Parallelogram *)));

        //写入数据，将已有物体的地址存入数组中
        vector<const Sphere *> sphereVector;
        vector<const Parallelogram *> parallelogramVector;

        for (size_t i = 0; i < objectListSize; i++) {
            switch (objectList[i].first) {
                case PrimitiveType::SPHERE:
                    sphereVector.push_back(&dev_spheres[objectList[i].second]);
                    break;
                case PrimitiveType::PARALLELOGRAM:
                    parallelogramVector.push_back(&dev_parallelograms[objectList[i].second]);
                    break;
                default:;
            }
        }

        //拷贝到显存
        cudaCheckError(hipMemcpy(dev_hittablePDFSpheres, sphereVector.data(), sphereCount * sizeof(const Sphere *), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_hittablePDFParallelograms, parallelogramVector.data(), parallelogramCount * sizeof(const Parallelogram *), hipMemcpyHostToDevice));

        this->hittablePDFSphereCount = sphereCount;
        this->hittablePDFParallelogramCount = parallelogramCount;
        this->isDirectSample = true;
    }

    void Renderer::commitSceneData(const Rough * roughMaterials, Uint32 roughMaterialCount,
                         const Metal * metalMaterials, Uint32 metalMaterialCount,
                         const Dielectric * dielectricMaterials, Uint32 dielectricMaterialCount,
                         const DiffuseLight * diffuseLightMaterials, Uint32 diffuseLightMaterialCount,
                         const Sphere * spheres, Uint32 sphereCount,
                         const Triangle * triangles, Uint32 triangleCount,
                         const Parallelogram * parallelograms, Uint32 parallelogramCount,
                         const Box * boxs, Uint32 boxCount,
                         const Transform * transforms, Uint32 transformCount)
    {
        SDL_Log("Commit data...");
        SDL_Log("Constructing BVH...");

        //构建BVH，仅添加没有被变换的图元
#define _constructVector(className, arrayName) \
        vector<className> arrayName##Vector;\
        for (size_t i = 0; i < arrayName##Count; i++) {\
            if (!arrayName##s[i].isTransformed) arrayName##Vector.push_back(arrayName##s[i]);\
        }
        //============
        _constructVector(Sphere, sphere);
        _constructVector(Triangle, triangle);
        _constructVector(Parallelogram, parallelogram);
        _constructVector(Box, box);
        //============
#undef _constructVector
        const vector<Transform> transformVector(transforms, transforms + transformCount);

        //先利用vector的返回值传递接收数组，再转换为指针
        const auto ret = BVHTree::constructBVHTree(sphereVector, triangleVector, parallelogramVector, boxVector, transformVector);

        const auto tree = ret.first.data();
        const auto treeIndexArray = ret.second.data();

        const size_t treeSize = ret.first.size() * sizeof(BVHTree::BVHTreeNode);
        const size_t treeIndexArraySize = ret.second.size() * sizeof(pair<PrimitiveType, size_t>);

        //分配BVH树显存
        cudaCheckError(hipMalloc(&dev_tree, treeSize));
        cudaCheckError(hipMalloc(&dev_treeIndexArray, treeIndexArraySize));

        //拷贝BVH数据
        cudaCheckError(hipMemcpy(dev_tree, tree, treeSize, hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_treeIndexArray, treeIndexArray, treeIndexArraySize, hipMemcpyHostToDevice));

        //============

        //分配场景数据显存
        SDL_Log("Construction complete, allocating VRAM...");

        cudaCheckError(hipMalloc(&dev_roughMaterials, roughMaterialCount * sizeof(Rough)));
        cudaCheckError(hipMalloc(&dev_metalMaterials, metalMaterialCount * sizeof(Metal)));
        cudaCheckError(hipMalloc(&dev_dielectricMaterials, dielectricMaterialCount * sizeof(Dielectric)));
        cudaCheckError(hipMalloc(&dev_diffuseLightMaterials, diffuseLightMaterialCount * sizeof(DiffuseLight)));
        cudaCheckError(hipMalloc(&dev_spheres, sphereCount * sizeof(Sphere)));
        cudaCheckError(hipMalloc(&dev_triangles, triangleCount * sizeof(Triangle)));
        cudaCheckError(hipMalloc(&dev_parallelograms, parallelogramCount * sizeof(Parallelogram)));
        cudaCheckError(hipMalloc(&dev_boxes, boxCount * sizeof(Box)));
        cudaCheckError(hipMalloc(&dev_transforms, transformCount * sizeof(Transform)));

        SDL_Log("VRAM allocation complete.");

        //拷贝场景数据
        SDL_Log("Copying data...");

        cudaCheckError(hipMemcpy(dev_roughMaterials, roughMaterials, roughMaterialCount * sizeof(Rough), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_metalMaterials, metalMaterials, metalMaterialCount * sizeof(Metal), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_dielectricMaterials, dielectricMaterials, dielectricMaterialCount * sizeof(Dielectric), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_diffuseLightMaterials, diffuseLightMaterials, diffuseLightMaterialCount * sizeof(DiffuseLight), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_spheres, spheres, sphereCount * sizeof(Sphere), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_triangles, triangles, triangleCount * sizeof(Triangle), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_parallelograms, parallelograms, parallelogramCount * sizeof(Parallelogram), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_boxes, boxs, boxCount * sizeof(Box), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(dev_transforms, transforms, transformCount * sizeof(Transform), hipMemcpyHostToDevice));

        SDL_Log("Data copying complete.");

        //============

        this->devPointerAvailable = true;
    }

    void Renderer::freeSceneData() {
        //============

        //释放场景数据显存
        SDL_Log("Free VRAM...");

        cudaCheckError(hipFree(dev_roughMaterials));
        cudaCheckError(hipFree(dev_metalMaterials));
        cudaCheckError(hipFree(dev_dielectricMaterials));
        cudaCheckError(hipFree(dev_diffuseLightMaterials));
        cudaCheckError(hipFree(dev_spheres));
        cudaCheckError(hipFree(dev_triangles));
        cudaCheckError(hipFree(dev_parallelograms));
        cudaCheckError(hipFree(dev_boxes));
        cudaCheckError(hipFree(dev_transforms));

        //============

        //释放BVH树显存
        cudaCheckError(hipFree(dev_tree));
        cudaCheckError(hipFree(dev_treeIndexArray));

        //释放采样物体指针数组显存
        cudaCheckError(hipFree(dev_hittablePDFSpheres));
        cudaCheckError(hipFree(dev_hittablePDFParallelograms));
        this->isDirectSample = false;

        SDL_Log("VRAM free success.");
        this->devPointerAvailable = false;
    }

    void Renderer::renderFrame(const Camera * cam, SDL_Window * window, bool isPrintInfo) const {
        //检查参数
        if (!devPointerAvailable) {
            SDL_LogError(SDL_LOG_CATEGORY_ERROR, "Device pointers not available!");
            return;
        }

        SDL_Surface * surface = SDL_GetWindowSurface(window);
        if (surface == nullptr) {
            SDL_LogError(SDL_LOG_CATEGORY_ERROR, "Surface pointer is nullptr!");
            return;
        }

        if (isPrintInfo) {
            SDL_Log("Pixel format: %s", SDL_GetPixelFormatName(surface->format->format));
        }

        //创建设备端像素缓冲区
        const size_t pixelCount = surface->w * surface->h;
        Uint32 * dev_pixelBuffer;
        cudaCheckError(hipMalloc(&dev_pixelBuffer, pixelCount * sizeof(Uint32)));

        //拷贝设备端渲染器和相机对象
        Renderer * dev_renderer;
        Camera * dev_camera;
        cudaCheckError(hipMalloc(&dev_renderer, sizeof(Renderer)));
        cudaCheckError(hipMemcpy(dev_renderer, this, sizeof(Renderer), hipMemcpyHostToDevice));
        cudaCheckError(hipMalloc(&dev_camera, sizeof(Camera)));
        cudaCheckError(hipMemcpy(dev_camera, cam, sizeof(Camera), hipMemcpyHostToDevice));

        //将整个屏幕划分为16x16的网格，每个网格对应一个block
        const dim3 blocks(cam->windowWidth % 16 == 0 ? cam->windowWidth / 16 : cam->windowWidth / 16 + 1,
                          cam->windowHeight % 16 == 0 ? cam->windowHeight / 16 : cam->windowHeight / 16 + 1, 1);
        const dim3 threads(16, 16, 1);

        //初始化线程随机数生成器
        hiprandState * dev_stateArray;
        cudaCheckError(hipMalloc(&dev_stateArray, pixelCount * sizeof(hiprandState)));
        initThreadRandom<<<blocks, threads>>>(dev_stateArray);
        cudaCheckError(hipDeviceSynchronize());

        //记录事件
        hipEvent_t start, finish;
        cudaCheckError(hipEventCreate(&start));
        cudaCheckError(hipEventCreate(&finish));
        cudaCheckError(hipEventRecord(start, nullptr));

        //启动渲染
        if (isPrintInfo) {
            SDL_Log("Rendering...");
        }
        render<<<blocks, threads>>>(dev_renderer, dev_camera, dev_pixelBuffer, dev_stateArray);
        //cudaCheckError(hipDeviceSynchronize());

        //统计用时
        cudaCheckError(hipEventRecord(finish, nullptr));
        cudaCheckError(hipEventSynchronize(finish));
        float timeUsed;
        cudaCheckError(hipEventElapsedTime(&timeUsed, start, finish));
        if (isPrintInfo) {
            SDL_Log("Render complete. Time: %.2fms", timeUsed);
        }
        cudaCheckError(hipEventDestroy(start));
        cudaCheckError(hipEventDestroy(finish));

        //拷贝缓冲区颜色到主机并显示，主机启动核函数后异步执行，必须等待核函数执行完毕
#define USING_BUFFER
#ifdef USING_BUFFER
        SDL_Delay(100);
        auto * pixelBuffer = new Uint32 [pixelCount];
        cudaCheckError(hipMemcpy(pixelBuffer, dev_pixelBuffer, pixelCount * sizeof(Uint32), hipMemcpyDeviceToHost));

        //将缓冲区中的颜色拷贝到surface
        memcpy(surface->pixels, pixelBuffer, pixelCount * sizeof(Uint32));
        SDL_UpdateWindowSurface(window);
        delete[] pixelBuffer;
#else
        //可以直接复制到surface
        cudaCheckError(hipMemcpy(surface->pixels, dev_pixelBuffer, pixelCount * sizeof(Uint32), hipMemcpyDeviceToHost));
        SDL_UpdateWindowSurface(window);
#endif

        //释放临时资源
        cudaCheckError(hipFree(dev_pixelBuffer));
        cudaCheckError(hipFree(dev_camera));
        cudaCheckError(hipFree(dev_renderer));

        //保存渲染结果
        if (isPrintInfo) {
            SDL_CheckErrorInt(IMG_SavePNG(surface, "../files/output.png"), "Save PNG");
        }
    }

    void Renderer::printDeviceInfo() {
        SDL_Log("Querying devices...");

        hipDeviceProp_t prop {};
        int deviceCount;

        cudaCheckError(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            SDL_LogError(SDL_LOG_CATEGORY_ERROR, "No CUDA device detected!");
            exit(EXIT_FAILURE);
        }

        SDL_Log("CUDA Version: %d.%d",  CUDART_VERSION / 1000, (CUDART_VERSION % 1000) / 10);
        SDL_Log("==================================================");
        for (int i = 0; i < deviceCount; i++) {
            cudaCheckError(hipGetDeviceProperties(&prop, i));
            SDL_Log("Device name: %s", prop.name);
            SDL_Log("Compute capability: %d.%d", prop.major, prop.minor);

            SDL_Log("Total global memory: %.2f MB", (double)prop.totalGlobalMem / (1024 * 1024));
            SDL_Log("Shared memory per block: %.2f KB", (double)prop.sharedMemPerBlock / 1024);
            SDL_Log("Reserved shared memory per block: %.2f KB", (double)prop.reservedSharedMemPerBlock / 1024);
            SDL_Log("Memory bus width: %d bits", prop.memoryBusWidth);
            SDL_Log("L2 cache size: %.2f KB", (double)prop.l2CacheSize / 1024);
            SDL_Log("Total constant memory: %.2f KB", (double)prop.totalConstMem / 1024);

            SDL_Log("Clock rate: %d kHz", prop.clockRate);
            SDL_Log("Memory clock rate: %d kHz", prop.memoryClockRate);

            SDL_Log("Registers per block: %d", prop.regsPerBlock);
            SDL_Log("Max threads per block: %d", prop.maxThreadsPerBlock);

            SDL_Log("Warp size: %d", prop.warpSize);
            SDL_Log("Multiprocessor count: %d", prop.multiProcessorCount);
            SDL_Log("Max blocks per multiprocessor: %d", prop.maxBlocksPerMultiProcessor);
            SDL_Log("Max threads per multiprocessor: %d", prop.maxThreadsPerMultiProcessor);
            SDL_Log("Shared memory per multiprocessor: %.2f KB", (double)prop.sharedMemPerMultiprocessor / 1024);
            SDL_Log("Registers per multiprocessor: %d", prop.regsPerMultiprocessor);

            SDL_Log("Max threads dimensions: (%d, %d, %d)",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
            SDL_Log("Max grid size: (%d, %d, %d)",
                    prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
            SDL_Log("Max texture dimensions: (%d, %d, %d)",
                    prop.maxTexture1D, prop.maxTexture2D[0], prop.maxTexture2D[1]);
            SDL_Log("Max surface dimensions: (%d, %d, %d)",
                    prop.maxSurface1D, prop.maxSurface2D[0], prop.maxSurface2D[1]);
            SDL_Log("==================================================");
        }
    }
}