#include "hip/hip_runtime.h"
#include <util/Matrix.cuh>

namespace renderer {
    __host__ __device__ Matrix Matrix::transpose() const {
        //创建新矩阵，元素均为0，排除原有数据干扰
        Matrix ret;
        //将第i行变为第i列
        for (size_t i = 1; i <= row; i++) { //操作的矩阵可能为列向量，不固定为5
            for (size_t j = 1; j <= col; j++) {
                ret.data[j][i] = data[i][j];
            }
        }
        return ret;
    }

    __host__ __device__ Matrix Matrix::operator*(const Matrix &right) const {
        //创建新的矩阵，行数为左矩阵的行数，列数为右矩阵的列数
        Matrix ret(row, right.col);
        //矩阵乘法，逐个元素赋值
        for (size_t i = 1; i <= ret.row; i++) {
            for (size_t j = 1; j <= ret.col; j++) {
                double sum = 0.0;
                for (size_t n = 1; n <= col; n++) {
                    sum += data[i][n] * right.data[n][j];
                }
                ret.data[i][j] = sum;
            }
        }
        return ret;
    }

    //只有4x4方阵而非列向量能够调用inverse
    __host__ __device__ Matrix Matrix::inverse() const {
        /*
         * 构造同阶单位矩阵，并将其合并到参数矩阵的右侧
         * 由于Matrix类的data限定大小为4x4，因此使用临时数组代替对象
         */
        double operateMatrix[5][9] {};
        for (size_t i = 1; i < 5; i++) {
            //将原矩阵数据填入左半部分
            for (size_t j = 1; j < 5; j++) {
                operateMatrix[i][j] = data[i][j];
            }
            //将右半部分设置为单位矩阵
            operateMatrix[i][4 + i] = 1.0;
        }

        //对合并后的矩阵进行两次消元
        if (eliminateBottomElements(operateMatrix) != 0 || eliminateTopElements(operateMatrix) != 0) {
            //矩阵不满秩，无法求逆
            return *this;
        }

        //操作矩阵的右半部分即为所求
        Matrix ret;
        for (size_t i = 1; i < 5; i++) {
            for (size_t j = 1; j < 5; j++) {
                ret.data[i][j] = operateMatrix[i][4 + j];
            }
        }
        return ret;
    }

    __host__ __device__ int Matrix::eliminateBottomElements(double matrixData[5][9]) {
        //前向消元
        for (size_t i = 1; i < 5; i++) {
            //主元选择
            double main = abs(matrixData[i][i]);
            size_t maxRow = i;
            //选取当前行及以下行最大的主元
            for (size_t p = i + 1; p < 5; p++) {
                if (abs(matrixData[p][i]) > main) {
                    main = abs(matrixData[p][i]);
                    maxRow = p;
                }
            }

            //检查最大主元是否为零
            if (floatValueNearZero(main)) {
                return 1;
            }

            if (maxRow != i) {
                //交换第maxRow行和第i行
                double tmp[9] {};
                for (size_t j = 1; j < 9; j++) {
                    tmp[j] = matrixData[maxRow][j];
                }
                for (size_t j = 1; j < 9; j++) {
                    matrixData[maxRow][j] = matrixData[i][j];
                }
                for (size_t j = 1; j < 9; j++) {
                    matrixData[i][j] = tmp[j];
                }
            }

            //操作当前行（i）的下方所有行
            for (size_t j = i + 1; j < 5; j++) {
                const double factor = matrixData[j][i] / matrixData[i][i];
                //操作一行
                for (size_t k = i; k < 9; k++) {
                    matrixData[j][k] -= factor * matrixData[i][k];
                }
            }
        }
        return 0;
    }

    __host__ __device__ int Matrix::eliminateTopElements(double matrixData[5][9]) {
        for (size_t i = 4; i >= 1; i--) {
            if (floatValueNearZero(matrixData[i][i])) {
                if (!floatValueNearZero(matrixData[i][8])) {
                    return 1; //无解
                } else {
                    return 2; //有无数个解
                }
            }

            //计算归一化系数。增广部分的所有元素都需要进行缩放
            double factor = 1 / matrixData[i][i];
            for (size_t p = i; p < 9; p++) {
                matrixData[i][p] *= factor;
            }
            for (size_t j = i - 1; j >= 1; j--) {
                factor = matrixData[j][i];
                for (size_t k = j; k < 9; k++) {
                    matrixData[j][k] -= factor * matrixData[i][k];
                }
            }
        }
        return 0;
    }

    __host__ Matrix Matrix::constructShiftMatrix(const std::array<double, 3> &shift) {
        return Matrix(4, 4, {
                1.0, 0.0, 0.0, shift[0],
                0.0, 1.0, 0.0, shift[1],
                0.0, 0.0, 1.0, shift[2],
                0.0, 0.0, 0.0, 1.0
        });
    }

    Matrix Matrix::constructScaleMatrix(const std::array<double, 3> &scale) {
        return Matrix(4, 4, {
                scale[0], 0.0, 0.0, 0.0,
                0.0, scale[1], 0.0, 0.0,
                0.0, 0.0, scale[2], 0.0,
                0.0, 0.0, 0.0, 1.0
        });
    }

    Matrix Matrix::constructRotateMatrix(double degree, int axis) {
        const double theta = degreeToRadian(degree);
        switch (axis) {
            case 0:
                return Matrix(4, 4, {
                        1.0, 0.0, 0.0, 0.0,
                        0.0, cos(theta), -sin(theta), 0.0,
                        0.0, sin(theta), cos(theta), 0.0,
                        0.0, 0.0, 0.0, 1.0
                });
            case 1:
                return Matrix(4, 4, {
                        cos(theta), 0.0, sin(theta), 0.0,
                        0.0, 1.0, 0.0, 0.0,
                        -sin(theta), 0.0, cos(theta), 0.0,
                        0.0, 0.0, 0.0, 1.0
                });
            case 2:
                return Matrix(4, 4, {
                        cos(theta), -sin(theta), 0.0, 0.0,
                        sin(theta), cos(theta), 0.0, 0.0,
                        0.0, 0.0, 1.0, 0.0,
                        0.0, 0.0, 0.0, 1.0
                });
            default:
                throw std::runtime_error("Invalid axis index!");
        }
    }

    Matrix Matrix::constructRotateMatrix(const std::array<double, 3> &rotate) {
        const auto mx = constructRotateMatrix(rotate[0], 0);
        const auto my = constructRotateMatrix(rotate[1], 1);
        const auto mz = constructRotateMatrix(rotate[2], 2);
        return mx * my * mz;
    }
}